#include "cuda_utils.hpp"

void* alloc_pinned_mem(size_t len)
{
    void* ptr;
    hipError_t status = hipHostMalloc(&ptr, len);
    if (status != hipSuccess)
    {
        ptr = NULL;
    }
    return ptr;
}

void free_pinned_mem(void* ptr)
{
    hipHostFree(ptr);
}