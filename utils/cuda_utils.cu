#include "cuda_utils.hpp"
#include "cuda_utils.cuh"

#define MAX_GPUS 16

void* alloc_pinned_mem(size_t len)
{
    void* ptr;
    hipError_t status = hipHostMalloc(&ptr, len);
    if (status != hipSuccess)
    {
        ptr = NULL;
    }
    return ptr;
}

void free_pinned_mem(void* ptr)
{
    hipHostFree(ptr);
}

void warmup_all_gpus()
{
    uint64_t *gpu_a[MAX_GPUS];
    uint64_t size = (1 << 20);

    int nDevices = 0;
    CHECKCUDAERR(hipGetDeviceCount(&nDevices));

#pragma omp parallel for num_threads(nDevices)
    for (uint32_t d = 0; d < nDevices; d++)
    {
        CHECKCUDAERR(hipSetDevice(d));
        CHECKCUDAERR(hipMalloc(&gpu_a[d], size * sizeof(uint64_t)));
    }
#pragma omp parallel for num_threads(nDevices)
    for (uint32_t d = 0; d < nDevices; d++)
    {
        CHECKCUDAERR(hipSetDevice(d));
        CHECKCUDAERR(hipFree(gpu_a[d]));
    }
}